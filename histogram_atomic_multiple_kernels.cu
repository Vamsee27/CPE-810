/*
   Lab2: Histogram using Atomic operations, shared memory

   Name: Vamsee Krishna Tunuguntla
   CWID: 20009051

   Constraints: Input vector<VecDim> values should be in between 0 to 1023, VecD
im size is inarbitrary, <BinNum> should not exceed 2^8
   Code has been implemented taking the instructions into consideration and using multiple kernels
   */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

__global__ void histo_kernel(unsigned int *input, long size, unsigned int *histo, int bin_num)
{
    // Shared memory to hold histogram bins
    __shared__ unsigned int s_histo[256];

    // Initialize shared memory bins to 0
    for (int i = threadIdx.x; i < bin_num; i += blockDim.x)
        s_histo[i] = 0;
    __syncthreads();

    // Calculate stride and thread index
    int stride = blockDim.x * gridDim.x;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < size)
    {
        // Calculate histogram bin for input value
        int bin = input[tid] / (1024 / bin_num);
        
        // Increment histogram bin using atomic operation
        atomicAdd(&(s_histo[bin]), 1);
        
        tid += stride;
    }
    __syncthreads();

    // Accumulate per-thread histogram bins into global histogram
    for (int i = threadIdx.x; i < bin_num; i += blockDim.x)
        atomicAdd(&(histo[i]), s_histo[i]);
}

int main(int argc, char *argv[])
{
    // Parse input arguments
    if (argc != 5)
    {
        printf("Usage: %s -i <BinNum> <VecDim> <BlockSize>\n", argv[0]);
        exit(1);
    }

    int bin_num = atoi(argv[2]);
    long vec_dim = atol(argv[3]);
    int block_size = atoi(argv[4]);

    // Allocate host memory
    unsigned int *h_input = (unsigned int *)malloc(vec_dim * sizeof(unsigned int));
    unsigned int *h_histo = (unsigned int *)calloc(bin_num, sizeof(unsigned int));

    // Generate random input vector
    for (int i = 0; i < vec_dim; i++)
        h_input[i] = rand() % 1024;

   /*
    // Print the input vector
    printf("Input vector:\n");
    for (int i = 0; i < vec_dim; i++) {
        printf("%d ", h_input[i]);
    }
    printf("\n");
    */

    // Allocate device memory
    unsigned int *d_input, *d_histo;
    hipMalloc((void **)&d_input, vec_dim * sizeof(unsigned int));
    hipMalloc((void **)&d_histo, bin_num * sizeof(unsigned int));

    // Copy host memory to device
    hipMemcpy(d_input, h_input, vec_dim * sizeof(unsigned int), hipMemcpyHostToDevice);

    // Calculate grid dimensions for the first kernel launch
    dim3 block_dim_1(block_size, 1, 1);
    dim3 grid_dim_1((vec_dim / 2 + block_size - 1) / block_size, 1, 1);

    // Invoke the first CUDA kernel
    histo_kernel<<<grid_dim_1, block_dim_1>>>(d_input, vec_dim / 2, d_histo, bin_num);

    // Calculate grid dimensions for the second kernel launch
    dim3 block_dim_2(block_size, 1, 1);
    dim3 grid_dim_2(((vec_dim - vec_dim / 2) + block_size - 1) / block_size, 1, 1);

    // Invoke the second CUDA kernel
    histo_kernel<<<grid_dim_2, block_dim_2>>>(d_input + vec_dim / 2, vec_dim - vec_dim / 2, d_histo, bin_num);

    // Copy results from device to host
    hipMemcpy(h_histo, d_histo, bin_num * sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Deallocate device memory
    hipFree(d_input);
    hipFree(d_histo);

    // Print histogram results
    for (int i = 0; i < bin_num; i++)
        printf("%d: %u\n", i, h_histo[i]);

   // Count total number of elements from all bins
   int total_elements = 0;
   for (int i = 0; i < bin_num; i++){
         total_elements += h_histo[i];
   }
   //printf("Vector Dimension (Total elements): %ld\n", vec_dim);
   printf("Total Elements in bins: %d\n", total_elements);

    // Deallocate host memory
    free(h_input);
    free(h_histo);

    return 0;
}
